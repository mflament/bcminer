
#include <hip/hip_runtime.h>
#define H_INTS 8
#define BLOCK_INTS 16
#define BUFFER_INTS 64

typedef unsigned int uint;

__constant__ uint DEFAULT_H[H_INTS] = { 0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A, 0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19 };

__constant__ uint K[64] = {
	  0x428A2F98, 0x71374491, 0xB5C0FBCF, 0xE9B5DBA5,
	  0x3956C25B, 0x59F111F1, 0x923F82A4, 0xAB1C5ED5,
	  0xD807AA98, 0x12835B01, 0x243185BE, 0x550C7DC3,
	  0x72BE5D74, 0x80DEB1FE, 0x9BDC06A7, 0xC19BF174,
	  0xE49B69C1, 0xEFBE4786, 0x0FC19DC6, 0x240CA1CC,
	  0x2DE92C6F, 0x4A7484AA, 0x5CB0A9DC, 0x76F988DA,
	  0x983E5152, 0xA831C66D, 0xB00327C8, 0xBF597FC7,
	  0xC6E00BF3, 0xD5A79147, 0x06CA6351, 0x14292967,
	  0x27B70A85, 0x2E1B2138, 0x4D2C6DFC, 0x53380D13,
	  0x650A7354, 0x766A0ABB, 0x81C2C92E, 0x92722C85,
	  0xA2BFE8A1, 0xA81A664B, 0xC24B8B70, 0xC76C51A3,
	  0xD192E819, 0xD6990624, 0xF40E3585, 0x106AA070,
	  0x19A4C116, 0x1E376C08, 0x2748774C, 0x34B0BCB5,
	  0x391C0CB3, 0x4ED8AA4A, 0x5B9CCA4F, 0x682E6FF3,
	  0x748F82EE, 0x78A5636F, 0x84C87814, 0x8CC70208,
	  0x90BEFFFA, 0xA4506CEB, 0xBEF9A3F7, 0xC67178F2
};

__device__ uint rotateRight(const uint value, const int bits)
{
	return (value >> bits) | (value << (32 - bits));
}

__device__ uint shiftRight(const uint value, const int bits)
{
	return (value >> bits);
}

__device__ uint ch(const uint x, const uint y, const uint z)
{
	return ((x & y) ^ ((~x) & z));
}

__device__ uint maj(const uint x, const uint y, const uint z)
{
	return ((x & y) ^ (x & z) ^ (y & z));
}

__device__ uint sigma0256(const uint x)
{
	return (rotateRight(x, 2) ^ rotateRight(x, 13) ^ rotateRight(x, 22));
}

__device__ uint sigma1256(const uint x)
{
	return (rotateRight(x, 6) ^ rotateRight(x, 11) ^ rotateRight(x, 25));
}

__device__ uint gamma0256(const uint x)
{
	return (rotateRight(x, 7) ^ rotateRight(x, 18) ^ shiftRight(x, 3));
}

__device__ uint gamma1256(const uint x)
{
	return (rotateRight(x, 17) ^ rotateRight(x, 19) ^ shiftRight(x, 10));
}

__device__ uint swap(const uint i)
{
	return ((i >> 24) & 0xff) | // move byte 3 to byte 0
		   ((i << 8) & 0xff0000) | // move byte 1 to byte 2
		   ((i >> 8) & 0xff00) | // move byte 2 to byte 1
		   ((i << 24) & 0xff000000); // byte 0 to byte 3
}

__device__ void clear(uint* array, const int start, const int end)
{
	for (uint i = start; i < end; i++)
		array[i] = 0;
}

__device__ void processBlock(uint* wb, uint* target)
{
	int i;
	uint a = target[0];
	uint b = target[1];
	uint c = target[2];
	uint d = target[3];
	uint e = target[4];
	uint f = target[5];
	uint g = target[6];
	uint h = target[7];
	uint T1, T2;

	for (i = 16; i < BUFFER_INTS; i++) {
		wb[i] = gamma1256(wb[i - 2]) + wb[i - 7] + gamma0256(wb[i - 15]) + wb[i - 16];
	}

	for (i = 0; i < BUFFER_INTS; i++) {
		T1 = h + sigma1256(e) + ch(e, f, g) + K[i] + wb[i];
		T2 = sigma0256(a) + maj(a, b, c);

		h = g;
		g = f;
		f = e;
		e = d + T1;
		d = c;
		c = b;
		b = a;
		a = T1 + T2;
	}

	target[0] += a;
	target[1] += b;
	target[2] += c;
	target[3] += d;
	target[4] += e;
	target[5] += f;
	target[6] += g;
	target[7] += h;
}

__device__ bool test_hash(const int hMaskOffset, const uint hMask, uint* hash)
{
	int offset = hMaskOffset;
	uint sum = hash[offset] & hMask;
	for (int i = offset + 1; i < H_INTS; i++) {
		sum |= hash[i];
	}
	return sum == 0;
}

__device__ void hash_block(const uint header[3], const uint midstate[H_INTS], uint nonce, uint* workBuffer, uint* hash)
{
	workBuffer[0] = header[0]; // last int of merkel root
	workBuffer[1] = header[1]; // time
	workBuffer[2] = header[2]; // nbits
	workBuffer[3] = swap(nonce); // nonce
	//Padding
	workBuffer[4] = 0x80000000;
	clear(workBuffer, 5, BLOCK_INTS - 1);
	// size (in bits) = 80 * 8 =  640
	workBuffer[BLOCK_INTS - 1] = 640;

    for (int i = 0; i < H_INTS; i++) hash[i] = midstate[i];
	processBlock(workBuffer, hash);

	for (int i = 0; i < H_INTS; i++) workBuffer[i] = hash[i];
	workBuffer[H_INTS] = 0x80000000; // padding
	clear(workBuffer, 9, BLOCK_INTS - 1);
	workBuffer[BLOCK_INTS - 1] = 256; // size (16 * 16)

	for (int i = 0; i < H_INTS; i++) hash[i] = DEFAULT_H[i];
	processBlock(workBuffer, hash);
}

/**
* Data layout
* [0] last int of merkel root
* [1] time
* [2] nBits
* [3+8] midState
* [11] hMastOffset (int)
* [12] hMask
*/
extern "C" __global__ void mine(uint *globalData, const uint baseNonce, const uint nonceCount, uint *result)
{
    __shared__ uint header[3];
    __shared__ uint midstate[8];
    extern __shared__ int localMatches[];

    int hMaskOffset = (int)globalData[11];
    uint hMask = globalData[12];

    uint workBuffer[BUFFER_INTS];
    uint hash[H_INTS];

    uint groupCount = gridDim.x;
    uint groupNonces = nonceCount / groupCount;
    uint groupId = blockIdx.x;
    uint localId = threadIdx.x;
    uint groupSize = blockDim.x;

    uint startNonce = baseNonce + groupId * groupNonces + localId;

    for (int i = localId; i < 3; i += groupSize) header[i] = globalData[i];
    for (int i = localId; i < 8; i += groupSize) midstate[i] = globalData[3 + i];
    localMatches[localId] = -1;
    __syncthreads();

//     if (groupId == 0 && localId == 0)
//         printf("baseNonce: %u, nonceCount: %u result[0]=%u\n", baseNonce, nonceCount, result[0]);

    uint nonce;
    for (uint i = 0; i < groupNonces; i += groupSize) {
        nonce = startNonce + i;
        hash_block(header, midstate, nonce, workBuffer, hash);
        if (test_hash(hMaskOffset, hMask, hash)) {
            // printf("nonce matched: group id %u local id: %u nonce: %u\n", groupId, localId, nonce);
            localMatches[localId] = localId;
        }
        __syncthreads();
        for(int s = groupSize/2; s > 0; s >>= 1) {
            if (localId < s) {
                localMatches[localId] = localMatches[localId] < 0 ? localMatches[localId + s] : localMatches[localId];
            }
            __syncthreads();
        }
        if (localId == 0 && localMatches[0] >= 0) {
            // printf("matched by local id %d\n", localMatches[0]);
            result[0] = 1;
            result[1] = nonce + localMatches[0];
        }
        __threadfence();
        if (result[0] != 0) {
            // printf("nonce matched: group id %d local id: %d result: %d\n", groupId, localId, result[0]);
            break;
        }
    }
}